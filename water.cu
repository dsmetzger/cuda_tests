/*
*/

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cube(float * d_out, float * d_in){
	int idx = threadIdx.x;
	float f = d_in[idx];
	d_out[idx] = f * f * f;
}

int main(int argc, char ** argv) {

}
